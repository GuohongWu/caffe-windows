#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <cmath>

#include "caffe/layers/cosine_contrastive_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

	template <typename Dtype>
	void CosineContrastiveLossLayer<Dtype>::Forward_gpu(
		const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {

		if (this->add_weighted) {
			int total_nums = bottom[2]->num();
			int pos_count = 0;
			for (int i = 0; i < total_nums; ++i) {
				if (static_cast<int>(bottom[2]->cpu_data()[i]))
					++pos_count;
			}
			if (pos_count == 0 || pos_count == total_nums) {
				this->pos_weight_ = this->neg_weight_ = Dtype(1);
			}
			else {
				this->pos_weight_ = Dtype(0.5) * Dtype(total_nums) / Dtype(pos_count); // n/(2*k)
				this->neg_weight_ = Dtype(0.5) * Dtype(total_nums) / Dtype(total_nums - pos_count); // n/(2*(n-k))
			}
		}

		Dtype margin = this->layer_param_.contrastive_loss_param().margin();
		Dtype loss(0.0);
		const int channels = bottom[0]->channels();
		for (int i = 0; i < bottom[0]->num(); ++i) {
			Dtype dot_prod_out, x_len_out, y_len_out;
			caffe_gpu_dot(channels, bottom[0]->gpu_data() + (i*channels), bottom[1]->gpu_data() + (i*channels), &dot_prod_out);
			caffe_gpu_dot(channels, bottom[0]->gpu_data() + (i*channels), bottom[0]->gpu_data() + (i*channels), &x_len_out);
			caffe_gpu_dot(channels, bottom[1]->gpu_data() + (i*channels), bottom[1]->gpu_data() + (i*channels), &y_len_out);

			//this->dot_prod_.mutable_cpu_data()[i] = dot_out;
			this->x_len_.mutable_cpu_data()[i] = sqrt(x_len_out);
			this->y_len_.mutable_cpu_data()[i] = sqrt(y_len_out);

			Dtype cur_cosine_val = dot_prod_out / (this->x_len_.cpu_data()[i] + Dtype(1e-6)) / (this->y_len_.cpu_data()[i] + Dtype(1e-6));
			this->cos_theta_.mutable_cpu_data()[i] = cur_cosine_val;
			this->sin_theta_.mutable_cpu_data()[i] = sqrt(Dtype(1) - cur_cosine_val * cur_cosine_val);

			Dtype cos_theta_plus_m = cur_cosine_val * this->cos_m - this->sin_m * this->sin_theta_.cpu_data()[i];
			if (static_cast<int>(bottom[2]->cpu_data()[i])) {  // similar pairs
				if (cur_cosine_val + this->cos_m >= Dtype(0))
					loss += this->pos_weight_ * (Dtype(1) - cos_theta_plus_m);
				else
					loss += this->pos_weight_ * (Dtype(3) + cos_theta_plus_m);
			}
			else {  // dissimilar pairs
				if (cur_cosine_val + this->cos_m >= Dtype(0))
					loss += this->neg_weight_ * std::max(margin, cos_theta_plus_m);
				else
					loss += this->neg_weight_ * std::max(margin, Dtype(-2) - cos_theta_plus_m);
			}
		}
		loss = loss / static_cast<Dtype>(bottom[0]->num());
		top[0]->mutable_cpu_data()[0] = loss;
	}

	template <typename Dtype>
	void CosineContrastiveLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

		Dtype margin = this->layer_param_.contrastive_loss_param().margin();
		int num = bottom[0]->num();
		int channels = bottom[0]->channels();
		const Dtype alpha = top[0]->cpu_diff()[0] / static_cast<Dtype>(num);

		if (propagate_down[0]) {
			Dtype* bout = bottom[0]->mutable_gpu_diff();
			for (int j = 0; j < num; ++j) {
				Dtype cur_cos_theta = this->cos_theta_.cpu_data()[j];
				Dtype cur_sin_theta = this->sin_theta_.cpu_data()[j];
				Dtype diff_to_cosTheta = Dtype(0);

				if (static_cast<int>(bottom[2]->cpu_data()[j])) {  // similar pairs
					diff_to_cosTheta = -this->pos_weight_ * (this->cos_m + this->sin_m * cur_cos_theta / (cur_sin_theta + Dtype(1e-6)));
					if (cur_cos_theta + this->cos_m < Dtype(0))
						diff_to_cosTheta = -diff_to_cosTheta;

					caffe_copy(channels, bottom[1]->gpu_data() + (j * channels), bout + (j * channels));
					caffe_gpu_axpby(channels,
						-alpha * diff_to_cosTheta * cur_cos_theta / (this->x_len_.cpu_data()[j] * this->x_len_.cpu_data()[j] + Dtype(1e-6)),
						bottom[0]->gpu_data() + (j * channels),
						alpha * diff_to_cosTheta / (this->x_len_.cpu_data()[j] * this->y_len_.cpu_data()[j] + Dtype(1e-6)),
						bout + (j * channels));
				}
				else {
					Dtype mdist = this->cos_m * cur_cos_theta - this->sin_m * cur_sin_theta;
					if (cur_cos_theta + this->cos_m < Dtype(0))
						mdist = Dtype(-2) - mdist;

					if (mdist > margin) {
						diff_to_cosTheta = this->neg_weight_ * (this->cos_m + this->sin_m * cur_cos_theta / (cur_sin_theta + Dtype(1e-6)));
						if (cur_cos_theta + this->cos_m < Dtype(0))
							diff_to_cosTheta = -diff_to_cosTheta;

						caffe_copy(channels, bottom[1]->gpu_data() + (j * channels), bout + (j * channels));
						caffe_gpu_axpby(channels,
							-alpha * diff_to_cosTheta * cur_cos_theta / (this->x_len_.cpu_data()[j] * this->x_len_.cpu_data()[j] + Dtype(1e-6)),
							bottom[0]->gpu_data() + (j * channels),
							alpha * diff_to_cosTheta / (this->x_len_.cpu_data()[j] * this->y_len_.cpu_data()[j] + Dtype(1e-6)),
							bout + (j * channels));
					}
					else {
						caffe_gpu_set(channels, Dtype(0), bout + (j*channels));
					}
				}
			}
		}

		if (propagate_down[1]) {
			Dtype* bout = bottom[1]->mutable_gpu_diff();
			for (int j = 0; j < num; ++j) {
				Dtype cur_cos_theta = this->cos_theta_.cpu_data()[j];
				Dtype cur_sin_theta = this->sin_theta_.cpu_data()[j];
				Dtype diff_to_cosTheta = Dtype(0);

				if (static_cast<int>(bottom[2]->cpu_data()[j])) {  // similar pairs
					diff_to_cosTheta = -this->pos_weight_ * (this->cos_m + this->sin_m * cur_cos_theta / (cur_sin_theta + Dtype(1e-6)));
					if (cur_cos_theta + this->cos_m < Dtype(0))
						diff_to_cosTheta = -diff_to_cosTheta;

					caffe_copy(channels, bottom[1]->gpu_data() + (j * channels), bout + (j * channels));
					caffe_gpu_axpby(channels,
						alpha * diff_to_cosTheta / (this->x_len_.cpu_data()[j] * this->y_len_.cpu_data()[j] + Dtype(1e-6)),
						bottom[0]->gpu_data() + (j * channels),
						-alpha * diff_to_cosTheta * cur_cos_theta / (this->y_len_.cpu_data()[j] * this->y_len_.cpu_data()[j] + Dtype(1e-6)),
						bout + (j * channels));
				}
				else {
					Dtype mdist = this->cos_m * cur_cos_theta - this->sin_m * cur_sin_theta;
					if (cur_cos_theta + this->cos_m < Dtype(0))
						mdist = Dtype(-2) - mdist;

					if (mdist > margin) {
						diff_to_cosTheta = this->neg_weight_ * (this->cos_m + this->sin_m * cur_cos_theta / (cur_sin_theta + Dtype(1e-6)));
						if (cur_cos_theta + this->cos_m < Dtype(0))
							diff_to_cosTheta = -diff_to_cosTheta;

						caffe_copy(channels, bottom[1]->gpu_data() + (j * channels), bout + (j * channels));
						caffe_gpu_axpby(channels,
							alpha * diff_to_cosTheta / (this->x_len_.cpu_data()[j] * this->y_len_.cpu_data()[j] + Dtype(1e-6)),
							bottom[0]->gpu_data() + (j * channels),
							-alpha * diff_to_cosTheta * cur_cos_theta / (this->y_len_.cpu_data()[j] * this->y_len_.cpu_data()[j] + Dtype(1e-6)),
							bout + (j * channels));
					}
					else {
						caffe_gpu_set(channels, Dtype(0), bout + (j*channels));
					}
				}
			}
		}
	}

	INSTANTIATE_LAYER_GPU_FUNCS(CosineContrastiveLossLayer);

}  // namespace caffe
